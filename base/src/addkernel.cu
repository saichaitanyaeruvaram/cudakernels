#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "kernels.h"

__global__ void add(const Npp8u* src1, const Npp8u* src2, Npp8u* dst, int step, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int offset = y * step + x;
	dst[offset] = src1[offset] + src2[offset];
}

__global__ void add_32(const Npp8u* src1, const Npp8u* src2, Npp8u* dst, int step, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	for (auto i = 0; i < 64; i++)
	{
		int offset = y * step + x + i;
		dst[offset] = src1[offset] + src2[offset];
	}
}

void launchAddKernel(const Npp8u* src1, const Npp8u* src2, Npp8u* dst, int step, NppiSize size, hipStream_t stream, std::string method)
{
	if (method == ADD_BASIC)
	{
		dim3 block(64, 64);
		dim3 grid((size.width + block.x - 1) / block.x, (size.height + block.y - 1) / block.y);
		add << <grid, block, 0, stream >> > (src1, src2, dst, step, size.width, size.height);
	}
	else if (method == ADD_32)
	{ 
		auto width = size.width >> 6;
		dim3 block(64, 64);
		dim3 grid((width + block.x - 1) / block.x, (size.height + block.y - 1) / block.y);
		add_32 << <grid, block, 0, stream >> > (src1, src2, dst, step, width, size.height);
	}
}