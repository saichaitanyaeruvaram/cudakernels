#include "hip/hip_runtime.h"
#include "kernels.h"

#define BLOCK_SIZE 1024
#define UINT32_BLOCK_STEP_2 64 // 16X4
#define BLOCK_SIZE_4 256
//
//__global__ void rgbtohsvtorgb(const Npp8u* R, const Npp8u* G, const Npp8u* B, Npp8u* Rn, Npp8u* Bn, Npp8u* Gn)
//{
//	Npp32f nNormalizedR = (Npp32f)R * 0.003921569F; // / 255.0F
//	Npp32f nNormalizedG = (Npp32f)G * 0.003921569F;
//	Npp32f nNormalizedB = (Npp32f)B * 0.003921569F;
//	Npp32f nS;
//	Npp32f nH;
//	// Value
//	Npp32f nV = fmaxf(nNormalizedR, nNormalizedG);
//	nV = fmaxf(nV, nNormalizedB);
//	// Saturation
//	Npp32f nTemp = fminf(nNormalizedR, nNormalizedG);
//	nTemp = fminf(nTemp, nNormalizedB);
//	Npp32f nDivisor = nV - nTemp;
//	if (nV == 0.0F) // achromatics case
//	{
//		nS = 0.0F;
//		nH = 0.0F;
//	}
//	else // chromatics case
//		nS = nDivisor / nV;
//	// Hue:
//	Npp32f nCr = (nV - nNormalizedR) / nDivisor;
//	Npp32f nCg = (nV - nNormalizedG) / nDivisor;
//	Npp32f nCb = (nV - nNormalizedB) / nDivisor;
//	if (nNormalizedR == nV)
//		nH = nCb - nCg;
//	else if (nNormalizedG == nV)
//		nH = 2.0F + nCr - nCb;
//	else if (nNormalizedB == nV)
//		nH = 4.0F + nCg - nCr;
//	nH = nH * 0.166667F; // / 6.0F       
//	if (nH < 0.0F)
//		nH = nH + 1.0F;
//	H = (Npp8u)(nH * 255.0F);
//	S = (Npp8u)(nS * 255.0F);
//	V = (Npp8u)(nV * 255.0F);
//
//	Npp32f nNormalizedH = (Npp32f)H * 0.003921569F; // / 255.0F
//	Npp32f nNormalizedS = (Npp32f)S * 0.003921569F;
//	Npp32f nNormalizedV = (Npp32f)V * 0.003921569F;
//	Npp32f nR;
//	Npp32f nG;
//	Npp32f nB;
//	if (nNormalizedS == 0.0F)
//	{
//		nR = nG = nB = nNormalizedV;
//	}
//	else
//	{
//		if (nNormalizedH == 1.0F)
//			nNormalizedH = 0.0F;
//		else
//			nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
//	}
//	Npp32f nI = floorf(nNormalizedH);
//	Npp32f nF = nNormalizedH - nI;
//	Npp32f nM = nNormalizedV * (1.0F - nNormalizedS);
//	Npp32f nN = nNormalizedV * (1.0F - nNormalizedS * nF);
//	Npp32f nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
//	if (nI == 0.0F)
//	{
//		nR = nNormalizedV; nG = nK; nB = nM;
//	}
//	else if (nI == 1.0F)
//	{
//		nR = nN; nG = nNormalizedV; nB = nM;
//	}
//	else if (nI == 2.0F)
//	{
//		nR = nM; nG = nNormalizedV; nB = nK;
//	}
//	else if (nI == 3.0F)
//	{
//		nR = nM; nG = nN; nB = nNormalizedV;
//	}
//	else if (nI == 4.0F)
//	{
//		nR = nK; nG = nM; nB = nNormalizedV;
//	}
//	else if (nI == 5.0F)
//	{
//		nR = nNormalizedV; nG = nM; nB = nN;
//	}
//	R = (Npp8u)(nR * 255.0F);
//	G = (Npp8u)(nG * 255.0F);
//	B = (Npp8u)(nB * 255.0F);
//
//}
//


#define CLAMP_255(x) x < 0 ? 0 : (x > 255 ? 255 : x)

// https://en.wikipedia.org/wiki/YUV#Y%E2%80%B2UV420p_(and_Y%E2%80%B2V12_or_YV12)_to_RGB888_conversion

//#define YUV_TO_RGB( Y, U, V, R, G, B )                      \
//    do                                                      \
//    {                                                       \
//		int rTmp = Y + (1.370705 * V);                      \
//        int gTmp = Y - (0.698001 * V) - (0.337633 * U);     \
//        int bTmp = Y + (1.732446 * U);						\
//        R = CLAMP_255(rTmp);                                \
//        G = CLAMP_255(gTmp);                                \
//        B = CLAMP_255(bTmp);                                \
//	} while (0)

#define YUV_TO_RGB( Y, U, V, R, G, B )                      \
    do                                                      \
    {                                                       \
		float rTmp = Y + __fmul_rd (1.370705, V);                      \
        float gTmp = Y - __fmul_rd (0.698001, V) - __fmul_rd (0.337633, U);     \
        float bTmp = Y + __fmul_rd (1.732446, U);						\
        R = CLAMP_255(rTmp);                                \
        G = CLAMP_255(gTmp);                                \
        B = CLAMP_255(bTmp);                                \
	} while (0)


__global__ void yuv420torgb(const uchar4* Y, const uint32_t* U, const uint32_t* V, uchar4* r, uchar4* g, uchar4* b, int width, int height, int step_y, int step_uv)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int offset = y * step_y + x;

	__shared__ uint32_t u_data[BLOCK_SIZE_4];
	__shared__ uint32_t v_data[BLOCK_SIZE_4];
		
	// read for every 4 frames once            
	if (threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0)
	{
		// 16*threadIdx.y*0.5 + threadIdx.x*0.5
		auto uvThreadOffset = (threadIdx.y << 3) + (threadIdx.x >> 1);
		auto uvOffset = (y >> 1) * (step_uv)+(x >> 1);
		u_data[uvThreadOffset] = U[uvOffset];
		v_data[uvThreadOffset] = V[uvOffset];
	}

	__syncthreads();

	// 32x32x4 y, r, g, b values
	// 16x16x4 u, v values

	auto u_data_uint8 = reinterpret_cast<uint8_t*>(u_data);
	auto v_data_uint8 = reinterpret_cast<uint8_t*>(v_data);
	   
	auto uvThreadOffset = (threadIdx.y >> 1)*UINT32_BLOCK_STEP_2 + (threadIdx.x << 1);
	int u_value = u_data_uint8[uvThreadOffset] - 128;
	int v_value = v_data_uint8[uvThreadOffset] - 128;
		
	YUV_TO_RGB(Y[offset].x, u_value, v_value, r[offset].x, g[offset].x, b[offset].x);
	YUV_TO_RGB(Y[offset].y, u_value, v_value, r[offset].y, g[offset].y, b[offset].y);

	uvThreadOffset += 1;
	u_value = u_data_uint8[uvThreadOffset] - 128;
	v_value = v_data_uint8[uvThreadOffset] - 128;
	YUV_TO_RGB(Y[offset].z, u_value, v_value, r[offset].z, g[offset].z, b[offset].z);
	YUV_TO_RGB(Y[offset].w, u_value, v_value, r[offset].w, g[offset].w, b[offset].w);
}

__global__ void yuv420torgb_plain2(const uint8_t* Y, const uint8_t* U, const uint8_t* V, uint8_t* r, uint8_t* g, uint8_t* b, int width, int height, int step_y, int step_uv)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int offset = y * step_y + x;
	auto uvOffset = (y >> 1) * (step_uv)+(x >> 1);

	int u_value = U[uvOffset] - 128;
	int v_value = V[uvOffset] - 128;

	YUV_TO_RGB(Y[offset], u_value, v_value, r[offset], g[offset], b[offset]);	
}

__global__ void yuv420torgb_plain(const uchar4* Y, const uint8_t* U, const uint8_t* V, uchar4* r, uchar4* g, uchar4* b, int width, int height, int step_y, int step_uv)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int offset = y * step_y + x;
	auto uvOffset = (y >> 1) * (step_uv)+(x << 1);
	
	int u_value = U[uvOffset] - 128;
	int v_value = V[uvOffset] - 128;

	YUV_TO_RGB(Y[offset].x, u_value, v_value, r[offset].x, g[offset].x, b[offset].x);
	YUV_TO_RGB(Y[offset].y, u_value, v_value, r[offset].y, g[offset].y, b[offset].y);

	uvOffset += 1;
	u_value = U[uvOffset] - 128;
	v_value = V[uvOffset] - 128;
	YUV_TO_RGB(Y[offset].z, u_value, v_value, r[offset].z, g[offset].z, b[offset].z);
	YUV_TO_RGB(Y[offset].w, u_value, v_value, r[offset].w, g[offset].w, b[offset].w);
}

void launch_yuv420torgb(const Npp8u* Y, const Npp8u* U, const Npp8u* V, Npp8u* R, Npp8u* G, Npp8u* B, int step_y, int step_uv, NppiSize size, hipStream_t stream, std::string method)
{	
	if (method == "plain")
	{
		auto width = size.width >> 2;
		step_y = step_y >> 2;
		step_uv = step_uv >> 2;
		dim3 block(32, 32);
		dim3 grid((width + block.x - 1) / block.x, (size.height + block.y - 1) / block.y);
		yuv420torgb_plain << <grid, block, 0, stream >> > (reinterpret_cast<const uchar4*>(Y), reinterpret_cast<const uint8_t*>(U), reinterpret_cast<const uint8_t*>(V), reinterpret_cast<uchar4*>(R), reinterpret_cast<uchar4*>(G), reinterpret_cast<uchar4*>(B), width, size.height, step_y, step_uv);
	}
	else if (method == "plain2")
	{
		auto width = size.width;
		dim3 block(32, 32);
		dim3 grid((width + block.x - 1) / block.x, (size.height + block.y - 1) / block.y);
		yuv420torgb_plain2 << <grid, block, 0, stream >> > (reinterpret_cast<const uint8_t*>(Y), reinterpret_cast<const uint8_t*>(U), reinterpret_cast<const uint8_t*>(V), reinterpret_cast<uint8_t*>(R), reinterpret_cast<uint8_t*>(G), reinterpret_cast<uint8_t*>(B), size.width, size.height, step_y, step_uv);
	}
	else
	{
		auto width = size.width >> 2;
		step_y = step_y >> 2;
		step_uv = step_uv >> 2;
		dim3 block(32, 32);
		dim3 grid((width + block.x - 1) / block.x, (size.height + block.y - 1) / block.y);
		yuv420torgb << <grid, block, 0, stream >> > (reinterpret_cast<const uchar4*>(Y), reinterpret_cast<const uint32_t*>(U), reinterpret_cast<const uint32_t*>(V), reinterpret_cast<uchar4*>(R), reinterpret_cast<uchar4*>(G), reinterpret_cast<uchar4*>(B), width, size.height, step_y, step_uv);
	}
}